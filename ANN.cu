#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "./headers/ANN.cuh"


/**
 *  Feedforward Kernels
 */


/**
 * Performs y = x*M
 * 
 * @param ans       [output of product]
 * @param vector    [the vector]
 * @param matrix    [the matrix]
 * @param w_row_dim [length of vector]
 * @param w_col_dim [length of columns]
 */
__global__
void callDotVectorToMatrix(float *ans, float *vector, float *matrix, int w_row_dim, int w_col_dim) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < w_col_dim) {

		float sum = 0.0;
		for(int row = 0; row < w_row_dim; row++) {
			sum += vector[row] * matrix[row * w_col_dim + i];
		}

		ans[i] = sum;

		i += blockDim.x * gridDim.x;
	}
}



void dotVectorToMatrix(unsigned int maxBlocks, unsigned int threadsPerBlock, float *ans, float *vector, float *matrix, int x_col_dim, int w_row_dim, int w_col_dim, hipStream_t stream) {

	callDotVectorToMatrix<<<maxBlocks, threadsPerBlock, 0, stream>>>(ans, vector, matrix, w_row_dim, w_col_dim);
}



/**
 * Add two vectors elementwise and performs the sigmoid on the sum.
 * This kernel combines the addition kernel and the sigmoid kernel.
 * 
 * @param output [output of the operation]
 * @param input  [input array from input_layer]
 * @param arr    [array to be added to input]
 * @param size   [size of one of the arrays; they should be the same size]
 */
__global__
void callAddVectors(float *output, float *input, float *arr, int size) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {

		// Get sum
		float sum = input[i] + arr[i];

		// Perform sigmoid
		float sigmoid = 1.0 / (1.0 + exp(-sum)); 
		output[i] = sigmoid;

		i += blockDim.x * gridDim.x;
	}
}


void addVectors(unsigned int maxBlocks, unsigned int threadsPerBlock, float *output, float *input, float *arr, int size, hipStream_t stream) {

	callAddVectors<<<maxBlocks, threadsPerBlock, 0, stream>>>(output, input, arr, size);
}





/**
 *  Backpropogate Kernels and helper functions
 */

/**
 * Calculate the deltas for a given layer.
 * d_i = d_{i-1} * W^T (hadamard) sigmoiddx(input_layer)
 * 
 * @param ans       [Output of the operation]
 * @param vector    [delta vector]
 * @param matrix    [matrix of weights]
 * @param input     [vector from input_layer]
 * @param w_row_dim [number of rows in matrix]
 * @param w_col_dim [number of columns in matrix]
 */
__global__
void callCalculateDeltas(float *ans, float *vector, float *matrix, float *input, int w_row_dim, int w_col_dim) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < w_row_dim) {

		// Compute delta_{i+1} * W^T
		
		float sum = 0.0;
		for (int col = 0; col < w_col_dim; col++) {
			sum += vector[col] * matrix[i * w_col_dim + col];
		}

		// Compute the sigmoid of input vector.
		float temp = 1.0 / (1.0 + exp(-input[i]));
		float sigmoid_dx = (1 - temp) * temp;

		// Compute Hadamard product. 
		ans[i] = sum * sigmoid_dx;

		i += blockDim.x * gridDim.x;
	}
}

void calculateDeltas(unsigned int maxBlocks, 
	unsigned int threadsPerBlock, 
	float *ans, float *vector, 
	float *matrix, float *input, int x_col_dim, 
	int w_row_dim, int w_col_dim, hipStream_t stream) {

	callCalculateDeltas<<<maxBlocks, threadsPerBlock, 0, stream>>>(ans, vector, matrix, input, w_row_dim, w_col_dim);
}



__global__
/**
 * Dots vector1 and vector2 and sets the bias gradient.
 * 
 * @param ans      [Output of dot product; weight gradient]
 * @param vector1  [vector from output_layer]
 * @param vector2  [vector from deltas]
 * @param delta_b  [bias gradient]
 * @param col_dim1 [size of vector1]
 * @param col_dim2 [size of vector2]
 */
void callDotVectorTransposeToVector(float *ans, float *vector1, float *vector2, float *delta_b, int col_dim1, int col_dim2) {

	unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
	unsigned int x = threadIdx.x;
	int v1_idx, v2_idx;
	float v2_temp;

	// Use shared memory for fast access of vector2. 
	extern __shared__ float v2[];

	while(x < col_dim2) {

		v2[x] = vector2[x];
		x += blockDim.x;
	}

	__syncthreads();

	while(i < col_dim1 * col_dim2) {

		v1_idx = (i / col_dim2) % col_dim1;
		v2_idx = i % col_dim2;
		v2_temp = v2[v2_idx];

		// Coalesced global memory write
		ans[i] = vector1[v1_idx] * v2_temp;

		if (i < col_dim2) {
			delta_b[i] = v2_temp;
		}

		i += blockDim.x * gridDim.x;
	}
}


void dotVectorTransposeToVector(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *ans, float *vector1, 
	float *vector2, float *delta_b,
	int col_dim1, int col_dim2, hipStream_t stream) {

	callDotVectorTransposeToVector<<<maxBlocks, threadsPerBlock, col_dim2 * sizeof(float), stream>>>(ans, vector1, vector2, delta_b, col_dim1, col_dim2);
}




/**
 *  Gradient Descent kernels
 */

/**
 *  Updates the bias with bias gradient
 *
 * gamme is the learning rate
 * delta_bias is the gradient
 * size is the size of the vectors.
 */
__global__
void callUpdateBias(float *bias, float gamma, float *delta_bias, int size) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {
		bias[i] -= gamma * delta_bias[i];

		i += blockDim.x * gridDim.x;
	}
}



void updateBias(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *bias, float gamma, 
	float *delta_bias, int size, hipStream_t stream) {

	callUpdateBias<<<maxBlocks, threadsPerBlock, 0, stream>>>(bias, gamma, delta_bias, size);
}

/**
 * Updates the weights with the gradient
 *
 * weights are the weights
 * gamme is the learning rate
 * delta_weights are the weight gradients
 * row_dim is the row dimension
 * col_dim is the column dimension
 * alpha is the regularization rate
 */
__global__
void callUpdateWeights(float *weights, float gamma, float *delta_weights, int row_dim, int col_dim, float alpha) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < row_dim * col_dim) {

		float w = weights[i];
		weights[i] = w - gamma * delta_weights[i] - alpha * w; 

		i += blockDim.x * gridDim.x;
	}
}

void updateWeights(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *weights, float gamma, 
	float *delta_weights, int row_dim, 
	int col_dim, float alpha, hipStream_t stream) {

	callUpdateWeights<<<maxBlocks, threadsPerBlock, 0, stream>>>(weights, gamma, delta_weights, row_dim, col_dim, alpha);
}



/**
 * Calculate delta of last layer
 *
 * store output in ans
 * predicted is the vector containing the predictions
 * label is the true label
 * size is the size of the vector
 */
__global__
void callDelta(float *ans, float *predicted, int label, int size) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {
		
		if (i == (int) label) 
			ans[i] = predicted[i] - 1;
		else 
			ans[i] = predicted[i];

		i += gridDim.x * blockDim.x;
	}
}


void delta(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *ans, float *predicted, 
	float label, int size, hipStream_t stream) {
	
	callDelta<<<maxBlocks, threadsPerBlock, 0, stream>>>(ans, predicted, (int) label, size);
}




