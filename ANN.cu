#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include "./headers/ANN.cuh"


/**
 *  Feedforward Kernels
 */

__global__
void callDotVectorToMatrix(float *ans, float *vector, float *matrix, int w_row_dim, int w_col_dim) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < w_row_dim) {

		float x_elem = vector[i];

		for(int col = 0; col < w_col_dim; col++) {
			float addend = x_elem * matrix[i * w_col_dim + col];
			atomicAdd(&ans[col], addend);
		}

		i += blockDim.x * gridDim.x;
	}
}



void dotVectorToMatrix(unsigned int maxBlocks, unsigned int threadsPerBlock, float *ans, float *vector, float *matrix, int x_col_dim, int w_row_dim, int w_col_dim) {

	callDotVectorToMatrix<<<maxBlocks, threadsPerBlock>>>(ans, vector, matrix, w_row_dim, w_col_dim);
}




__global__
void callAddVectors(float *output, float *input, float *arr, int size) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {
		float sum = input[i] + arr[i];
		float sigmoid = 1.0 / (1.0 + exp(-sum)); 
		output[i] = sigmoid;

		i += blockDim.x * gridDim.x;
	}
}


void addVectors(unsigned int maxBlocks, unsigned int threadsPerBlock, float *output, float *input, float *arr, int size) {

	callAddVectors<<<maxBlocks, threadsPerBlock>>>(output, input, arr, size);
}



__global__
void callSigmoid(float *output, float *input, int size) {
	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {
		output[i] = 1.0 / (1.0 + exp(-input[i])); 

		i += blockDim.x * gridDim.x;
	}
}

void sigmoid(unsigned int maxBlocks, unsigned int threadsPerBlock, float *output, float *input, int size) {
	callSigmoid<<<maxBlocks, threadsPerBlock>>>(output, input, size);
}










/**
 *  Backpropogate Kernels and helper functions
 */

__global__
void callCalculateDeltas(float *ans, float *vector, float *matrix, float *input, int w_row_dim, int w_col_dim) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < w_row_dim) {

		// Compute delta_{i+1} * W^T
		float sum = 0;
		for (int col = 0; col < w_col_dim; col++) {
			sum += vector[col] * matrix[i * w_col_dim + col];
		}

		// Compute the sigmoid of input vector.
		float temp = 1.0 / (1.0 + exp(-input[i]));
		float sigmoid_dx = (1 - temp) * temp;

		// Compute Hadamard product. 
		ans[i] = sum * sigmoid_dx;

		i += blockDim.x * gridDim.x;
	}
}

void calculateDeltas(unsigned int maxBlocks, 
	unsigned int threadsPerBlock, 
	float *ans, float *vector, 
	float *matrix, float *input, int x_col_dim, 
	int w_row_dim, int w_col_dim) {

	callCalculateDeltas<<<maxBlocks, threadsPerBlock>>>(ans, vector, matrix, input, w_row_dim, w_col_dim);
}



__global__
void callDotVectorTransposeToVector(float *ans, float *vector1, float *vector2, int col_dim1, int col_dim2) {

	int i1 = threadIdx.x + blockDim.x * blockIdx.x;

	while (i1 < col_dim1) {
		for (int i2 = 0; i2 < col_dim2; i2++) {
			ans[i1 * col_dim2 + i2] = vector1[i1] * vector2[i2];
		}

		i1 += blockDim.x * gridDim.x;
	}
}


void dotVectorTransposeToVector(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *ans, float *vector1, 
	float *vector2, int col_dim1, 
	int col_dim2) {

	callDotVectorTransposeToVector<<<maxBlocks, threadsPerBlock>>>(ans, vector1, vector2, col_dim1, col_dim2);
}




/**
 *  Gradient Descent kernels
 */
__global__
void callUpdateBias(float *bias, float gamma, float *delta_bias, int size) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < size) {
		bias[i] -= gamma * delta_bias[i];

		i += blockDim.x * gridDim.x;
	}
}



void updateBias(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *bias, float gamma, 
	float *delta_bias, int size) {

	callUpdateBias<<<maxBlocks, threadsPerBlock>>>(bias, gamma, delta_bias, size);
}


__global__
void callUpdateWeights(float *weights, float gamma, float *delta_weights, int row_dim, int col_dim, float alpha) {

	int i = threadIdx.x + blockDim.x * blockIdx.x;

	while(i < row_dim) {
		for (int col = 0; col < col_dim; col++) {
			weights[i * col_dim + col] -= gamma * delta_weights[i * col_dim + col] - alpha * weights[i * col_dim + col]; 
		}

		i += blockDim.x * gridDim.x;
	}
}

void updateWeights(unsigned int maxBlocks, 
	unsigned int threadsPerBlock,
	float *weights, float gamma, 
	float *delta_weights, int row_dim, 
	int col_dim, float alpha) {

	callUpdateWeights<<<maxBlocks, threadsPerBlock>>>(weights, gamma, delta_weights, row_dim, col_dim, alpha);
}










